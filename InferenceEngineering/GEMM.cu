#include "hip/hip_runtime.h"
# GEMM优化
__global__
void matrix_mul(float *A,float *B,float *C,int const M,const int K,int const N){
 
        int row=blockIdx.y*blockDim.y+threadIdx.y;
        int col=blockIdx.x*blockDim.x+threadIdx.x;
 
        __shared__ float tile_A[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float tile_B[BLOCK_SIZE][BLOCK_SIZE];
 
        float sum=0;
        for(int stride=0;stride<=K/BLOCK_SIZE;++stride){
                int id_m=row*K+stride*BLOCK_SIZE+threadIdx.x;
                if(row<M&&stride*BLOCK_SIZE+threadIdx.x<K){
                        tile_A[threadIdx.y][threadIdx.x]=A[id_m];
                }
                __syncthreads();
 
                int id_n=(stride*BLOCK_SIZE+threadIdx.y)*N+col;
                if(col<N&&stride*BLOCK_SIZE+threadIdx.y<K){
                        tile_B[threadIdx.y][threadIdx.x]=B[id_n];
                }
                __syncthreads();
 
                for(int i=0;i<BLOCK_SIZE;++i){
                        sum+=tile_A[threadIdx.y][i]*tile_B[i][threadIdx.x];
                }
                __syncthreads();
        }
        if(row<M&&col<N){
                C[row*N+col]=sum;
        }
}
