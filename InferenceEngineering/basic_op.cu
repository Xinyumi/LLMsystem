#include "hip/hip_runtime.h"
# Conv
__global__
void conv2d(const float *input,const float *kernel,const int in_size,const int ker_size,const int stride,float *output){
 
        int row=blockIdx.y*blockDim.y+threadIdx.y;
        int col=blockIdx.x*blockDim.x+threadIdx.x;
 
        const int out_size=(in_size-ker_size)/stride+1;
 
        float sum=0;
        if(row<out_size&&col<out_size){
                for(int i=0;i<ker_size;++i){
                        for(int j=0;j<ker_size;++j){
                                sum+=input[(row+i)*in_size+(col+j)]*kernel[i*ker_size+j];
                        }
                }
                output[row*out_size+col]=sum;
        }
        __syncthreads();
}


#maxpool
__global__
void maxpool(float *input,const int in_size,const int ker_size,const int stride,float *output){
 
        int row=blockIdx.y*blockDim.y+threadIdx.y;
        int col=blockIdx.x*blockDim.x+threadIdx.x;
 
        int out_size=(in_size-ker_size)/stride+1;
 
        if(row<out_size&&col<out_size){
                float max=0;
                for(int i=0;i<ker_size;++i){
                        for(int j=0;j<ker_size;++j){
                                float curr=input[(row*stride+i)*in_size+(col*stride+j)];
                                max=max<curr?curr:max;
                        }
                }
                output[row*out_size+col]=max;
        }
}


#im2col
# im2col的原理主要就是将输入矩阵中，需要进行操作的每一个子矩阵转变为一个列向量，然后使用矩阵乘法来进行计算
__global__
void im2col(const float *input,const int in_size,const int ker_size,const int stride,float *output){
 
        int idy=blockIdx.y*blockDim.y+threadIdx.y;
        int idx=blockIdx.x*blockDim.x+threadIdx.x;
 
        int out_size=(in_size-ker_size)/stride+1;
        int width=out_size*out_size;
        int height=ker_size*ker_size;
 
        if(idx<width && idy<height){
                int row=idx/out_size;
                int col=idx%out_size;
 
                int input_row=row+(idy/ker_size);
                int input_col=col+(idy%ker_size);
 
                int input_index=input_row*in_size+input_col;
                output[idy*width+idx]=input[input_index];
        }
}

#
